#include "hip/hip_runtime.h"
#include "elastodynamics_cqbem_collocation_kernels.cuh"

#include "integrator_collocation.hpp"

#ifdef __HIPCC__
#include <hip/hip_runtime.h>
template <typename T> inline T exp(const T &val) { return thrust::exp(val); }
#else
#define __device__
#define __host__
template <typename T> inline T exp(const T &val) { return std::exp(val); }
#endif

using KernelScalarType = float;
using SingularityScalarType = double;

__device__ __host__ void compute_elastodynamic_B_trans_kernel_collocation(
    ComplexType *B_buffer, const ScalarType *V_buffer, const IntType *F_buffer, const ScalarType *N_buffer,
    Eigen::Index num_vertices, Eigen::Index num_faces, ScalarType _c1, ScalarType _c2, complex<double> _s,
    unsigned int gaussian_quadrature_order, unsigned int quadrature_subdivision, int i,
    const IntType *vertex_map_inverse
) {
    using ThrustComplexType = complex<ScalarType>;
    using ThrustMatrixX3c = Eigen::Matrix<ThrustComplexType, Eigen::Dynamic, 3, StorageOrder>;

    using KernelRowVector3s = Eigen::Matrix<KernelScalarType, 1, 3>;
    using SingularityRowVector3s = Eigen::Matrix<SingularityScalarType, 1, 3>;

    using KernelThrustComplexType = complex<KernelScalarType>;
    using KernelThrustMatrix3c = Eigen::Matrix<KernelThrustComplexType, 3, 3, StorageOrder>;

    using SingularityComplexType = complex<SingularityScalarType>;

    auto B = Eigen::Map<ThrustMatrixX3c>(reinterpret_cast<ThrustComplexType *>(B_buffer), num_vertices * 3, 3);
    auto V = Eigen::Map<const MatrixX3s>(V_buffer, num_vertices, (Eigen::Index)3);
    auto F = Eigen::Map<const MatrixX3i>(F_buffer, num_faces, (Eigen::Index)3);
    auto N = Eigen::Map<const MatrixX3s>(N_buffer, num_faces, (Eigen::Index)3);

    KernelThrustMatrix3c Bi = KernelThrustMatrix3c::Zero();

    const auto r_b_hat_ij = [&](const auto &r_hat, auto r, const auto &n, int i, int j) -> complex<decltype(r)> {
        using IntegrandScalarType = decltype(r);
        using IntegrandComplexType = complex<decltype(r)>;
        const IntegrandScalarType c1 = _c1, c2 = _c2;
        const IntegrandComplexType s = IntegrandComplexType(_s);

        constexpr auto kronecker_delta = [](int i, int j) -> IntegrandScalarType { return (i == j) ? 1.0 : 0.0; };
        const IntegrandComplexType exp_neg_rs_c1 = exp(-r * s / c1);
        const IntegrandComplexType exp_neg_rs_c2 = exp(-r * s / c2);

        return -(IntegrandScalarType)(1. / (4. * M_PI)) *
               (((r_hat[i] * n[j] + r_hat[j] * n[i]) / ((IntegrandScalarType)2. * r)) *
                    (((s * r / c2 + (IntegrandScalarType)1.) / (s * s)) * exp_neg_rs_c2 -
                     ((s * r / c1 + (IntegrandScalarType)1.) / (s * s)) * exp_neg_rs_c1) +
                kronecker_delta(i, j) * r_hat.dot(n) / r *
                    ((IntegrandScalarType)1. / (s * s) -
                     ((s * r / c2 + (IntegrandScalarType)1.) / (s * s)) * exp_neg_rs_c2));
    };

    const auto b_hat = [&](const auto &y, const auto &x, const auto &n) -> KernelThrustMatrix3c {
        KernelThrustMatrix3c mat;
        auto r_vec = y - x;
        const KernelRowVector3s r_hat = r_vec.stableNormalized();
        KernelScalarType r = r_vec.norm();
        for (int i = 0; i < 3; i++) {
            for (int j = 0; j <= i; j++) {
                mat(i, j) = r_b_hat_ij(r_hat, r, n, i, j) / r;
                if (i != j) mat(j, i) = mat(i, j);
            }
        }
        return mat;
    };

    const KernelRowVector3s x = V.row(i).cast<KernelScalarType>();
    for (Eigen::Index f_index = 0; f_index < num_faces; f_index++) {
        const Eigen::Index j1 = F(f_index, 0), j2 = F(f_index, 1), j3 = F(f_index, 2);
        bool has_singular_point = true;
        int singular_vert_index;
        if (i == j1 || (vertex_map_inverse != nullptr && vertex_map_inverse[i] == vertex_map_inverse[j1]))
            singular_vert_index = 0;
        else if (i == j2 || (vertex_map_inverse != nullptr && vertex_map_inverse[i] == vertex_map_inverse[j2]))
            singular_vert_index = 1;
        else if (i == j3 || (vertex_map_inverse != nullptr && vertex_map_inverse[i] == vertex_map_inverse[j3]))
            singular_vert_index = 2;
        else
            has_singular_point = false;

        if (!has_singular_point) {
            const KernelRowVector3s n = N.row(f_index).cast<KernelScalarType>();
            const KernelRowVector3s y1 = V.row(j1).cast<KernelScalarType>(), y2 = V.row(j2).cast<KernelScalarType>(),
                                    y3 = V.row(j3).cast<KernelScalarType>();
            Bi += integrate_collocation_nonsingular(
                [&](const auto &y) { return b_hat(y, x, n); }, y1, y2, y3, gaussian_quadrature_order,
                quadrature_subdivision
            );
        } else {
            const SingularityRowVector3s n = N.row(f_index).cast<SingularityScalarType>();
            const Eigen::Index j1 = F(f_index, singular_vert_index), j2 = F(f_index, (singular_vert_index + 1) % 3),
                               j3 = F(f_index, (singular_vert_index + 2) % 3);
            const SingularityRowVector3s y1 = V.row(j1).cast<SingularityScalarType>(),
                                         y2 = V.row(j2).cast<SingularityScalarType>(),
                                         y3 = V.row(j3).cast<SingularityScalarType>();

            integrate_collocation_weakly_singular<SingularityScalarType>(
                [&](auto func) { // update loop
                    for (int p = 0; p < 3; p++) {
                        for (int q = 0; q <= p; q++) {
                            SingularityComplexType elem = func(Eigen::Vector2i(p, q));
                            Bi(p, q) += elem;
                            if (p != q) Bi(q, p) += elem;
                        }
                    }
                },
                [&](const auto &y, const auto &r_hat, const auto &r,
                    const auto &indices) -> SingularityComplexType { // integrand
                    return r_b_hat_ij(r_hat, r, n, indices[0], indices[1]);
                },
                y1, y2, y3, gaussian_quadrature_order, quadrature_subdivision
            );
        }
    }

    B.middleRows<3>(3 * i) = Bi.cast<ThrustComplexType>();
}

#ifdef __HIPCC__
__global__ void compute_elastodynamic_B_trans_kernel_collocation_global_wrapper(
    ComplexType *B_buffer, const ScalarType *V_buffer, const IntType *F_buffer, const ScalarType *N_buffer,
    Eigen::Index num_vertices, Eigen::Index num_faces, ScalarType c1, ScalarType c2, complex<double> s,
    unsigned int gaussian_quadrature_order, unsigned int quadrature_subdivision, unsigned int thread_per_block,
    const IntType *vertex_map_inverse
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_vertices) {
        compute_elastodynamic_B_trans_kernel_collocation(
            B_buffer, V_buffer, F_buffer, N_buffer, num_vertices, num_faces, c1, c2, s, gaussian_quadrature_order,
            quadrature_subdivision, i, vertex_map_inverse
        );
    }
}
#endif
