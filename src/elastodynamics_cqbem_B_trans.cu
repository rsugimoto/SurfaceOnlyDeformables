#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#else
#define __device__
#define __host__
#endif
#include <thread>

#include "elastodynamics_cqbem_matrix_collection.hpp"

#include "elastodynamics_cqbem_collocation_kernels.cuh"
#include "elastodynamics_cqbem_galerkin_kernels.cuh"

void ElastodynamicsCQBEMMatrixCollection::compute_B_trans_matrices(
    const MatrixX3s &V, const MatrixX3s &N, std::vector<MatrixX3s> &B_trans, bool enable_cuda
) {
    const double R = std::pow(1e-10, 1. / (2. * max_time_history));
    const size_t L = max_time_history + 1;
    const std::complex<double> exp_i2pi_L = std::exp(std::complex<double>(0.0, 1.0) * (2. * M_PI) / (double)L);
    const auto gamma_func =
        multistep_method == BDF1 ? [](const std::complex<double> &s) -> std::complex<double> { return 1. - s; }
    : [](const std::complex<double> &s) -> std::complex<double> { return 1.5 - 2. * s + s * s / 2.; };

#ifdef __HIPCC__
    hipStream_t stream;
    ScalarType *dev_V, *dev_N;
    IntType *dev_F, *dev_vertex_map_inverse;
    ComplexType *dev_B;
    if (enable_cuda) {
        hipStreamCreate(&stream);

        hipMallocAsync((void **)&dev_V, sizeof(ScalarType) * V.size(), stream);
        hipMallocAsync((void **)&dev_F, sizeof(IntType) * F.size(), stream);
        hipMallocAsync((void **)&dev_N, sizeof(ScalarType) * N.size(), stream);
        hipMallocAsync((void **)&dev_B, sizeof(ComplexType) * num_vertices * 3 * 3, stream);

        hipMemcpyAsync(dev_V, V.data(), sizeof(ScalarType) * V.size(), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(dev_F, F.data(), sizeof(IntType) * F.size(), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(dev_N, N.data(), sizeof(ScalarType) * N.size(), hipMemcpyHostToDevice, stream);

        if (enable_traction_discontinuity) {
            hipMallocAsync((void **)&dev_vertex_map_inverse, sizeof(IntType) * vertex_map_inverse.size(), stream);
            hipMemcpyAsync(
                dev_vertex_map_inverse, vertex_map_inverse.data(), sizeof(IntType) * vertex_map_inverse.size(),
                hipMemcpyHostToDevice, stream
            );
        }
    }
#endif

    MatrixX3c B(num_vertices * 3, 3);
    std::vector<MatrixX3c> B_list;
    for (size_t l = 0; l < L / 2 + 1; l++) {
        std::cout << "Compute B_trans hat matrices: " << l + 1 << "/" << L / 2 + 1 << std::endl;
        const ComplexType s = ComplexType(gamma_func(R * std::pow(exp_i2pi_L, l)) / (double)dt);

        if (use_galerkin) {
            if (enable_cuda) {
#ifdef __HIPCC__
                MatrixXc _B;
                std::thread cpu_thread([&]() {
                    _B = MatrixXc::Zero(num_vertices * 3, 3);
                    compute_elastodynamic_B_trans_kernel_galerkin_singular(
                        _B.data(), V.data(), F.data(), N.data(), num_vertices, F.rows(), c1, c2,
                        std::complex<SingularScalarType>(s), gaussian_quadrature_order, quadrature_subdivision,
                        enable_traction_discontinuity ? vertex_map_inverse.data() : nullptr
                    );
                });

                hipMemsetAsync(dev_B, 0, sizeof(ComplexType) * num_vertices * 3 * 3, stream);
                size_t _grid = (F.rows() + cuda_thread_per_block - 1) / cuda_thread_per_block;
                size_t _block = cuda_thread_per_block;
                dim3 grid(_grid, _grid);
                dim3 block(_block, _block);
                compute_elastodynamic_B_trans_kernel_galerkin_non_singular_global_wrapper<<<grid, block, 0, stream>>>(
                    dev_B, dev_V, dev_F, dev_N, num_vertices, F.rows(), c1, c2, s, gaussian_quadrature_order,
                    quadrature_subdivision, enable_traction_discontinuity ? dev_vertex_map_inverse : nullptr
                );
                hipError_t error = hipGetLastError();
                if (error != hipSuccess) std::cout << hipGetErrorString(error) << std::endl;
                hipMemcpyAsync(B.data(), dev_B, sizeof(ComplexType) * B.size(), hipMemcpyDeviceToHost, stream);
                hipStreamSynchronize(stream);

                cpu_thread.join();
                B += _B;
#endif
            } else {
                B.setZero();
#pragma omp parallel for
                for (Eigen::Index f1 = 0; f1 < F.rows(); f1++) {
                    for (Eigen::Index f2 = 0; f2 < F.rows(); f2++) {
                        compute_elastodynamic_B_trans_kernel_galerkin_non_singular(
                            B.data(), V.data(), F.data(), N.data(), num_vertices, F.rows(), c1, c2,
                            std::complex<NonSingularScalarType>(s), gaussian_quadrature_order, quadrature_subdivision,
                            f1, f2, enable_traction_discontinuity ? vertex_map_inverse.data() : nullptr
                        );
                    }
                }
                compute_elastodynamic_B_trans_kernel_galerkin_singular(
                    B.data(), V.data(), F.data(), N.data(), num_vertices, F.rows(), c1, c2,
                    std::complex<SingularScalarType>(s), gaussian_quadrature_order, quadrature_subdivision,
                    enable_traction_discontinuity ? vertex_map_inverse.data() : nullptr
                );
            }
        } else {
            if (enable_cuda) {
#ifdef __HIPCC__
                size_t grid = (num_vertices + cuda_thread_per_block - 1) / cuda_thread_per_block;
                size_t block = cuda_thread_per_block;
                compute_elastodynamic_B_trans_kernel_collocation_global_wrapper<<<grid, block, 0, stream>>>(
                    dev_B, dev_V, dev_F, dev_N, num_vertices, F.rows(), c1, c2, s, gaussian_quadrature_order,
                    quadrature_subdivision, cuda_thread_per_block,
                    enable_traction_discontinuity ? dev_vertex_map_inverse : nullptr
                );
                hipError_t error = hipGetLastError();
                if (error != hipSuccess) std::cout << hipGetErrorString(error) << std::endl;
                hipMemcpyAsync(B.data(), dev_B, sizeof(ComplexType) * B.size(), hipMemcpyDeviceToHost, stream);
                hipStreamSynchronize(stream);
#endif
            } else {
#pragma omp parallel for
                for (Eigen::Index i = 0; i < num_vertices; i++) {
                    compute_elastodynamic_B_trans_kernel_collocation(
                        B.data(), V.data(), F.data(), N.data(), num_vertices, F.rows(), c1, c2, s,
                        gaussian_quadrature_order, quadrature_subdivision, i,
                        enable_traction_discontinuity ? vertex_map_inverse.data() : nullptr
                    );
                }
            }
        }

        B_list.push_back(B);
    }

#ifdef __HIPCC__
    if (enable_cuda) {
        hipFreeAsync(dev_V, stream);
        hipFreeAsync(dev_F, stream);
        hipFreeAsync(dev_N, stream);
        hipFreeAsync(dev_B, stream);
        if (enable_traction_discontinuity) hipFreeAsync(dev_vertex_map_inverse, stream);

        hipStreamSynchronize(stream);
        hipStreamDestroy(stream);
    }
#endif

    for (size_t n = 0; n < L; n++) {
        std::cout << "Compute B_trans matrices: " << n + 1 << "/" << L << std::endl;
        MatrixX3s B = MatrixXs::Zero(num_vertices * 3, 3);
#pragma omp parallel for
        for (size_t l = 0; l < L / 2 + 1; l++) {
            const MatrixX3c &B_hat = B_list[l];
            MatrixX3s B_hat_real = (B_hat * ComplexType(std::pow(exp_i2pi_L, -(double)n * l))).real();
            if (l != 0 && (l != L / 2 || L % 2 == 1))
                B_hat_real += (B_hat.conjugate() * ComplexType(std::pow(exp_i2pi_L, -(double)n * (L - l)))).real();
#pragma omp critical
            B += B_hat_real;
        }
        B *= (ScalarType)(std::pow(R, -(double)n) / (double)L);

        B_trans.push_back(B);
    }
}
