#include "hip/hip_runtime.h"
#include "elastodynamics_cqbem_collocation_kernels.cuh"

#include "barycentric_coordinates.cuh"
#include "integrator_collocation.hpp"

#ifdef __HIPCC__
#include <hip/hip_runtime.h>
template <typename T> inline T exp(const T &val) { return thrust::exp(val); }
#else
#define __device__
#define __host__
template <typename T> inline T exp(const T &val) { return std::exp(val); }
#endif

using KernelScalarType = float;
using SingularityScalarType = double;

__device__ __host__ void compute_elastodynamic_G_kernel_collocation(
    ComplexType *U_buffer, const ScalarType *V_buffer, const IntType *F_buffer, Eigen::Index num_vertices,
    Eigen::Index num_faces, ScalarType _c1, ScalarType _c2, ScalarType _rho, complex<double> _s,
    unsigned int gaussian_quadrature_order, unsigned int quadrature_subdivision, int i,
    const IntType *vertex_map_inverse
) {
    using ThrustComplexType = complex<ScalarType>;
    using ThrustMatrixXc = Eigen::Matrix<ThrustComplexType, Eigen::Dynamic, Eigen::Dynamic, StorageOrder>;

    using KernelRowVector3s = Eigen::Matrix<KernelScalarType, 1, 3>;
    using SingularityRowVector3s = Eigen::Matrix<SingularityScalarType, 1, 3>;

    using KernelThrustComplexType = complex<KernelScalarType>;
    using KernelThrustVector3c = Eigen::Matrix<KernelThrustComplexType, 3, 1>;
    using KernelThrustMatrix3c = Eigen::Matrix<KernelThrustComplexType, 3, 3, StorageOrder>;

    using SingularityComplexType = complex<SingularityScalarType>;
    using SingularityVector3c = Eigen::Matrix<SingularityComplexType, 3, 1>;

    auto U =
        Eigen::Map<ThrustMatrixXc>(reinterpret_cast<ThrustComplexType *>(U_buffer), num_vertices * 3, num_vertices * 3);
    auto V = Eigen::Map<const MatrixX3s>(V_buffer, num_vertices, (Eigen::Index)3);
    auto F = Eigen::Map<const MatrixX3i>(F_buffer, num_faces, (Eigen::Index)3);

    U.middleRows<3>(3 * i).setZero();

    const auto r_u_hat_ij = [&](const auto &r_hat, auto r, int i, int j) -> complex<decltype(r)> {
        using IntegrandScalarType = decltype(r);
        using IntegrandComplexType = complex<decltype(r)>;
        const IntegrandScalarType c1 = _c1, c2 = _c2, rho = _rho;
        const IntegrandComplexType s = IntegrandComplexType(_s);

        constexpr auto kronecker_delta = [](int i, int j) -> IntegrandScalarType { return (i == j) ? 1.0 : 0.0; };
        const IntegrandComplexType exp_neg_rs_c1 = exp(-r * s / c1);
        const IntegrandComplexType exp_neg_rs_c2 = exp(-r * s / c2);

        return ((IntegrandScalarType)(1. / (4. * M_PI)) / rho) *
               ((((IntegrandScalarType)3. * r_hat[i] * r_hat[j] - kronecker_delta(i, j)) / (r * r)) *
                    (((s * r / c1 + (IntegrandScalarType)1.) / (s * s)) * exp_neg_rs_c1 -
                     ((s * r / c2 + (IntegrandScalarType)1.) / (s * s)) * exp_neg_rs_c2) +
                (r_hat[i] * r_hat[j]) * (exp_neg_rs_c1 / (c1 * c1) - exp_neg_rs_c2 / (c2 * c2)) +
                (kronecker_delta(i, j) / (c2 * c2)) * exp_neg_rs_c2);
    };

    const auto u_hat = [&](const auto &y, const auto &x) -> KernelThrustMatrix3c {
        KernelThrustMatrix3c mat;
        auto r_vec = y - x;
        const KernelRowVector3s r_hat = r_vec.stableNormalized();
        KernelScalarType r = r_vec.norm();
        for (int i = 0; i < 3; i++) {
            for (int j = 0; j <= i; j++) {
                mat(i, j) = r_u_hat_ij(r_hat, r, i, j) / r;
                if (i != j) mat(j, i) = mat(i, j);
            }
        }
        return mat;
    };

    const KernelRowVector3s x = V.row(i).cast<KernelScalarType>();
    for (Eigen::Index f_index = 0; f_index < num_faces; f_index++) {
        const Eigen::Index j1 = F(f_index, 0), j2 = F(f_index, 1), j3 = F(f_index, 2);
        bool has_singular_point = true;
        int singular_vert_index;
        if (i == j1 || (vertex_map_inverse != nullptr && vertex_map_inverse[i] == vertex_map_inverse[j1]))
            singular_vert_index = 0;
        else if (i == j2 || (vertex_map_inverse != nullptr && vertex_map_inverse[i] == vertex_map_inverse[j2]))
            singular_vert_index = 1;
        else if (i == j3 || (vertex_map_inverse != nullptr && vertex_map_inverse[i] == vertex_map_inverse[j3]))
            singular_vert_index = 2;
        else
            has_singular_point = false;

        if (!has_singular_point) {
            const KernelRowVector3s y1 = V.row(j1).cast<KernelScalarType>(), y2 = V.row(j2).cast<KernelScalarType>(),
                                    y3 = V.row(j3).cast<KernelScalarType>();
            const Eigen::Matrix<KernelThrustComplexType, 9, 3> mat = integrate_collocation_nonsingular(
                [&](const auto &y) {
                    KernelRowVector3s L;
                    barycentric_coordinates(y, y1, y2, y3, L);
                    L = L.cwiseMin(1.0).cwiseMax(0.0);
                    const auto _u_hat = u_hat(y, x);
                    Eigen::Matrix<KernelThrustComplexType, 9, 3> res;
                    for (int j = 0; j < 3; j++) res.middleRows<3>(3 * j) = _u_hat * L(j);
                    return res;
                },
                y1, y2, y3, gaussian_quadrature_order, quadrature_subdivision
            );
            U.block<3, 3>(3 * i, 3 * j1) += mat.block<3, 3>(0, 0).cast<ThrustComplexType>();
            U.block<3, 3>(3 * i, 3 * j2) += mat.block<3, 3>(3, 0).cast<ThrustComplexType>();
            U.block<3, 3>(3 * i, 3 * j3) += mat.block<3, 3>(6, 0).cast<ThrustComplexType>();
        } else {
            const Eigen::Index j1 = F(f_index, singular_vert_index), j2 = F(f_index, (singular_vert_index + 1) % 3),
                               j3 = F(f_index, (singular_vert_index + 2) % 3);
            const SingularityRowVector3s y1 = V.row(j1).cast<SingularityScalarType>(),
                                         y2 = V.row(j2).cast<SingularityScalarType>(),
                                         y3 = V.row(j3).cast<SingularityScalarType>();

            integrate_collocation_weakly_singular<SingularityScalarType>(
                [&](auto func) { // update loop
                    for (int p = 0; p < 3; p++) {
                        for (int q = 0; q <= p; q++) {
                            SingularityVector3c elems = func(Eigen::Vector2i(p, q));
                            U(3 * i + p, 3 * j1 + q) += elems(0);
                            U(3 * i + p, 3 * j2 + q) += elems(1);
                            U(3 * i + p, 3 * j3 + q) += elems(2);
                            if (p != q) {
                                U(3 * i + q, 3 * j1 + p) += elems(0);
                                U(3 * i + q, 3 * j2 + p) += elems(1);
                                U(3 * i + q, 3 * j3 + p) += elems(2);
                            }
                        }
                    }
                },
                [&](const auto &y, const auto &r_hat, const auto &r,
                    const auto &indices) -> SingularityVector3c { // integrand
                    SingularityRowVector3s L;
                    barycentric_coordinates(y, y1, y2, y3, L);
                    L = L.cwiseMin(1.0).cwiseMax(0.0);
                    return r_u_hat_ij(r_hat, r, indices[0], indices[1]) * L.cast<SingularityComplexType>();
                },
                y1, y2, y3, gaussian_quadrature_order, quadrature_subdivision
            );
        }
    }
}

#ifdef __HIPCC__
__global__ void compute_elastodynamic_G_kernel_collocation_global_wrapper(
    ComplexType *U_buffer, const ScalarType *V_buffer, const IntType *F_buffer, Eigen::Index num_vertices,
    Eigen::Index num_faces, ScalarType c1, ScalarType c2, ScalarType rho, complex<double> s,
    unsigned int gaussian_quadrature_order, unsigned int quadrature_subdivision, unsigned int thread_per_block,
    const IntType *vertex_map_inverse
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_vertices) {
        compute_elastodynamic_G_kernel_collocation(
            U_buffer, V_buffer, F_buffer, num_vertices, num_faces, c1, c2, rho, s, gaussian_quadrature_order,
            quadrature_subdivision, i, vertex_map_inverse
        );
    }
}
#endif