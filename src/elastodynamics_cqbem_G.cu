#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#else
#define __device__
#define __host__
#endif
#include <filesystem>
#include <thread>

#include "elastodynamics_cqbem_matrix_collection.hpp"
#include "matrix_io.hpp"

#include "elastodynamics_cqbem_collocation_kernels.cuh"
#include "elastodynamics_cqbem_galerkin_kernels.cuh"

#include "cuda_check_error.hpp"

void ElastodynamicsCQBEMMatrixCollection::compute_G_matrices(const MatrixX3s &V, bool enable_cuda) {
    const double R = std::pow(1e-10, 1. / (2. * max_time_history));
    const size_t L = max_time_history + 1;
    const std::complex<double> exp_i2pi_L = std::exp(std::complex<double>(0.0, 1.0) * (2. * M_PI) / (double)L);
    const auto gamma_func =
        multistep_method == BDF1 ? [](const std::complex<double> &s) -> std::complex<double> { return 1. - s; }
    : [](const std::complex<double> &s) -> std::complex<double> { return 1.5 - 2. * s + s * s / 2.; };

#ifdef __HIPCC__
    hipStream_t stream;
    ScalarType *dev_V;
    IntType *dev_F, *dev_vertex_map_inverse;
    ComplexType *dev_U;
    if (enable_cuda) {
        cuda_check_error([&]() { return hipStreamCreate(&stream); });
        cuda_check_error([&]() { return hipMallocAsync((void **)&dev_V, sizeof(ScalarType) * V.size(), stream); });
        cuda_check_error([&]() { return hipMallocAsync((void **)&dev_F, sizeof(IntType) * F.size(), stream); });
        cuda_check_error([&]() {
            return hipMallocAsync((void **)&dev_U, sizeof(ComplexType) * num_vertices * 3 * num_vertices * 3, stream);
        });

        cuda_check_error([&]() {
            return hipMemcpyAsync(dev_V, V.data(), sizeof(ScalarType) * V.size(), hipMemcpyHostToDevice, stream);
        });
        cuda_check_error([&]() {
            return hipMemcpyAsync(dev_F, F.data(), sizeof(IntType) * F.size(), hipMemcpyHostToDevice, stream);
        });

        if (enable_traction_discontinuity) {
            cuda_check_error([&]() {
                return hipMallocAsync(
                    (void **)&dev_vertex_map_inverse, sizeof(IntType) * vertex_map_inverse.size(), stream
                );
            });
            cuda_check_error([&]() {
                return hipMemcpyAsync(
                    dev_vertex_map_inverse, vertex_map_inverse.data(), sizeof(IntType) * vertex_map_inverse.size(),
                    hipMemcpyHostToDevice, stream
                );
            });
        }
    }
#endif

    MatrixXc U(num_vertices * 3, num_vertices * 3);
    for (size_t l = 0; l < L / 2 + 1; l++) {
        std::cout << "Compute G hat matrices: " << l + 1 << "/" << L / 2 + 1 << std::endl;
        const ComplexType s = ComplexType(gamma_func(R * std::pow(exp_i2pi_L, l)) / (double)dt);

        if (use_galerkin) {
            if (enable_cuda) {
#ifdef __HIPCC__
                MatrixXc _U;
                std::thread cpu_thread([&]() {
                    _U = MatrixXc::Zero(num_vertices * 3, num_vertices * 3);
                    compute_elastodynamic_G_kernel_galerkin_singular(
                        _U.data(), V.data(), F.data(), num_vertices, F.rows(), c1, c2, rho,
                        std::complex<SingularScalarType>(s), gaussian_quadrature_order, quadrature_subdivision,
                        enable_traction_discontinuity ? vertex_map_inverse.data() : nullptr
                    );
                });

                cuda_check_error([&]() {
                    return hipMemsetAsync(dev_U, 0, sizeof(ComplexType) * num_vertices * 3 * num_vertices * 3, stream);
                });
                size_t _grid = (F.rows() + cuda_thread_per_block - 1) / cuda_thread_per_block;
                size_t _block = cuda_thread_per_block;
                dim3 grid(_grid, _grid);
                dim3 block(_block, _block);

                compute_elastodynamic_G_kernel_galerkin_non_singular_global_wrapper<<<grid, block, 0, stream>>>(
                    dev_U, dev_V, dev_F, num_vertices, F.rows(), c1, c2, rho, s, gaussian_quadrature_order,
                    quadrature_subdivision, enable_traction_discontinuity ? dev_vertex_map_inverse : nullptr
                );
                cuda_check_last_error();
                cuda_check_error([&]() {
                    return hipMemcpyAsync(
                        U.data(), dev_U, sizeof(ComplexType) * U.size(), hipMemcpyDeviceToHost, stream
                    );
                });
                cuda_check_error([&]() { return hipStreamSynchronize(stream); });

                cpu_thread.join();
                U += _U;
#endif
            } else {
                U.setZero();
#pragma omp parallel for
                for (Eigen::Index f1 = 0; f1 < F.rows(); f1++) {
                    for (Eigen::Index f2 = 0; f2 < F.rows(); f2++) {
                        compute_elastodynamic_G_kernel_galerkin_non_singular(
                            U.data(), V.data(), F.data(), num_vertices, F.rows(), c1, c2, rho,
                            std::complex<NonSingularScalarType>(s), gaussian_quadrature_order, quadrature_subdivision,
                            f1, f2, enable_traction_discontinuity ? vertex_map_inverse.data() : nullptr
                        );
                    }
                }
                compute_elastodynamic_G_kernel_galerkin_singular(
                    U.data(), V.data(), F.data(), num_vertices, F.rows(), c1, c2, rho,
                    std::complex<SingularScalarType>(s), gaussian_quadrature_order, quadrature_subdivision,
                    enable_traction_discontinuity ? vertex_map_inverse.data() : nullptr
                );
            }
        } else {
            if (enable_cuda) {
#ifdef __HIPCC__
                size_t grid = (num_vertices + cuda_thread_per_block - 1) / cuda_thread_per_block;
                size_t block = cuda_thread_per_block;
                compute_elastodynamic_G_kernel_collocation_global_wrapper<<<grid, block, 0, stream>>>(
                    dev_U, dev_V, dev_F, num_vertices, F.rows(), c1, c2, rho, s, gaussian_quadrature_order,
                    quadrature_subdivision, cuda_thread_per_block,
                    enable_traction_discontinuity ? dev_vertex_map_inverse : nullptr
                );
                cuda_check_last_error();
                cuda_check_error([&]() {
                    return hipMemcpyAsync(
                        U.data(), dev_U, sizeof(ComplexType) * U.size(), hipMemcpyDeviceToHost, stream
                    );
                });
                cuda_check_error([&]() { return hipStreamSynchronize(stream); });
#endif
            } else {
#pragma omp parallel for
                for (Eigen::Index i = 0; i < num_vertices; i++) {
                    compute_elastodynamic_G_kernel_collocation(
                        U.data(), V.data(), F.data(), num_vertices, F.rows(), c1, c2, rho, s, gaussian_quadrature_order,
                        quadrature_subdivision, i, enable_traction_discontinuity ? vertex_map_inverse.data() : nullptr
                    );
                }
            }
        }

        Eigen::save_matrix(U, coeffs_folder_path + "/U_" + std::to_string(l) + ".mat");
    }

#ifdef __HIPCC__
    if (enable_cuda) {
        cuda_check_error([&]() { return hipFreeAsync(dev_V, stream); });
        cuda_check_error([&]() { return hipFreeAsync(dev_F, stream); });
        cuda_check_error([&]() { return hipFreeAsync(dev_U, stream); });
        if (enable_traction_discontinuity)
            cuda_check_error([&]() { return hipFreeAsync(dev_vertex_map_inverse, stream); });

        cuda_check_error([&]() { return hipStreamSynchronize(stream); });
        cuda_check_error([&]() { return hipStreamDestroy(stream); });
    }
#endif

    for (size_t n = 0; n < L; n++) {
        std::cout << "Compute G matrices: " << n + 1 << "/" << L << std::endl;
        MatrixXs _G = MatrixXs::Zero(num_vertices * 3, num_vertices * 3);
        for (size_t l = 0; l < L / 2 + 1; l++) {
            MatrixXc U;
            Eigen::load_matrix(U, coeffs_folder_path + "/U_" + std::to_string(l) + ".mat");
            _G += (U * ComplexType(std::pow(exp_i2pi_L, -(double)n * l))).real();
            if (l != 0 && (l != L / 2 || L % 2 == 1))
                _G += (U.conjugate() * ComplexType(std::pow(exp_i2pi_L, -(double)n * (L - l)))).real();
        }
        _G *= (ScalarType)(std::pow(R, -(double)n) / (double)L);

        Eigen::save_matrix(_G, coeffs_folder_path + "/G_" + std::to_string(n) + ".mat");
    }

    for (size_t l = 0; l < L / 2 + 1; l++)
        std::filesystem::remove(coeffs_folder_path + "/U_" + std::to_string(l) + ".mat");
}
